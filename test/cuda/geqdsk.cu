#include "hip/hip_runtime.h"
#include <fstream>
#include <string>
#include <cassert>

#include "geqdsk.hpp"
#include "files.hpp"
#include "cxxopts.hpp"
#include "types/scalar_field.hpp"

__global__ 
void kernel(Equilibrium eq, ScalarField psi, double *psi_sum, double *fpol_sum, int *idnum){
	*idnum = eq.idnum;

	*fpol_sum = 0;
	for (size_t i = 0; i < eq.fpol.size(); i++)
		*fpol_sum += eq.fpol[i];

	*psi_sum = 0;
	for(size_t i = 0; i < eq.psi.shape().first; i++)
		for(size_t j = 0; j < eq.psi.shape().second; j++)
			*psi_sum += psi(i, j);
}

int main(int argc, char* argv[]){
	
	cxxopts::options options("geqdsk", "Test geqdsk read");

	options.add_options()
		("file", "", cxxopts::value<std::string>())
		("h,help", "Show this help message");

	options.positional_help("<G-EQDSK input file>");
	options.parse_positional({"file"});

	try{
		auto result = options.parse(argc, argv);

		if (result.count("help")){
			std::cout << options.help() << std::endl;
			return 0;
		}

		Equilibrium eq = read_geqdsk(result["file"].as<std::string>().c_str());
		std::cout << eq.idnum << '\n';
		std::cout << eq.nx << '\n';
		std::cout << eq.ny << '\n';
		std::cout << eq.rdim << '\n';
		std::cout << eq.zdim << '\n';
		std::cout << eq.rcentr << '\n';
		std::cout << eq.rleft << '\n';
		std::cout << eq.zmid << '\n';
		std::cout << eq.rmagx << '\n';
		std::cout << eq.zmagx << '\n';
		std::cout << eq.simagx << '\n';
		std::cout << eq.sibdry << '\n';
		std::cout << eq.bcentr << '\n';
		std::cout << eq.cpasma << '\n';
		std::cout << eq.simagx << '\n';
		std::cout << eq.rmagx << '\n';
		std::cout << eq.zmagx << '\n';
		std::cout << eq.sibdry << '\n';

		dump("psi_from_geqdsk.dat", eq.psi, false);
		
		std::ofstream fo1("psi.dat");
		for (size_t j = 0; j < eq.ny; j++)
			for (size_t i = 0; i < eq.nx; i++){
				double r = eq.rleft + i * eq.rdim / eq.nx;
				double z = eq.zmid - eq.zdim / 2 + j * eq.zdim / eq.ny;
				fo1 << r << ' ' << z << ' ' << eq.psi(i, j) << '\n';
			}


		// Output limit
		std::ofstream fo("limit.dat");
		for (size_t i = 0; i < eq.nlim; i++)
			fo << eq.rlim[i] << ' ' << eq.zlim[i] << '\n';

		std::ofstream fo2("boundary.dat");
		for (size_t i = 0; i < eq.nlim; i++)
			fo2 << eq.rlim[i] << ' ' << eq.zlim[i] << '\n';
		
		double fpol_sum = 0;
		for (size_t i = 0; i < eq.fpol.size(); i++)
			fpol_sum += eq.fpol[i];

		double psi_sum = 0;	
		for(size_t i = 0; i < eq.psi.shape().first; i++)
			for(size_t j = 0; j < eq.psi.shape().second; j++)
				psi_sum += eq.psi(i, j);

		// Equilibrium dEq;
		// dEq.construct_in_host_for_device(eq);

		double mr_min = eq.rleft / eq.rdim;
		double mr_max = eq.rleft / eq.rdim + 1;
		double mz_min = (eq.zmid - 0.5 * eq.zdim) / eq.rdim;
		double mz_max = (eq.zmid + 0.5 * eq.zdim) / eq.rdim;

		ScalarField psi(eq.psi, mr_min, mr_max, mz_min, mz_max);

		double *d_psi_sum, *d_fpol_sum;
		int *d_idnum;

		hipMalloc(&d_psi_sum, sizeof(double));
		hipMalloc(&d_fpol_sum, sizeof(double));
		hipMalloc(&d_idnum, sizeof(int));

		kernel<<<1, 1>>>(eq, psi, d_psi_sum, d_fpol_sum, d_idnum);

		double h_psi_sum, h_fpol_sum;
		int h_idnum;

		hipMemcpy(&h_psi_sum, d_psi_sum, sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(&h_fpol_sum, d_fpol_sum, sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(&h_idnum, d_idnum, sizeof(int), hipMemcpyDeviceToHost);

		assert(fpol_sum == h_fpol_sum);
		assert(psi_sum == h_psi_sum);
		assert(h_idnum == eq.idnum);

		return 0;
	}catch(cxxopts::option_error const& e){
		std::cerr << e.what() << std::endl;
		return 1;
	}
}