#include "hip/hip_runtime.h"
#include <iostream>

#include "types/array.hpp"



__global__ void k_sum_array(int dArr[], size_t* n, int* s){
	Array<int> arr(dArr, *n);
	
	*s = 0;
	for(size_t i = 0; i < arr.size(); i++)
		*s += arr[i];
}

int main(){
	size_t n = 10;
	size_t* dn;
	int hArr[n];
	for(size_t i = 0; i < n; i++)
		hArr[i] = i;
	int* dArr;

	int hsum;
	int* dsum;
	hipMalloc(&dsum, sizeof(int));
	hipMalloc(&dn, sizeof(size_t));
	hipMalloc(&dArr, sizeof(int) * n);

	hipMemcpy(dArr, hArr, sizeof(int) * n, hipMemcpyHostToDevice);
	hipMemcpy(dn, &n, sizeof(size_t), hipMemcpyHostToDevice);

	k_sum_array<<<1, 1>>>(dArr, dn, dsum);

	hipMemcpy(&hsum, dsum, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dsum);
	hipFree(dArr);
	hipFree(dn);
	
	std::cout << hsum << '\n';

	return 0;
}