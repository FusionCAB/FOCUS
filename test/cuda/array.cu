#include "hip/hip_runtime.h"
#include <iostream>

#include "types/array.hpp"

__device__ Array<int> get_arr(size_t n){
	Array<int> arr(n);
	for(size_t i = 0; i < arr.size(); i++)
		arr[i] = i + 1;
	return arr; // move 
}

__global__ void k_sum_array(Array<int> arr, int* s){
	// Array<int> arr(dArr, *n); // construct for using the pointer from host and then don't try to deallocate
	Array<int> arr2 = get_arr(arr.size()); // move constructor and dealocate when leaving scope
	Array<int> arr3 = arr2; // copy constructor and dealocate when leaving scope
	Array<int> arr4 = arr; // copy constructor from another constructed from device, dealocate the new allocated when leaving scope

	*s = 0;
	for(size_t i = 0; i < arr.size(); i++)
		*s += arr[i] + arr2[i] + arr3[i] + arr4[i];
}


int main(){
		size_t n = 10;
		int hsum;
		int* dsum;
		
		//// Construct from C++ array
		// int raw_arr[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
		// Array<int> hArr(arr, n);

		// Construct from Array<T>, useful in order to use previously created interface
		Array<int> arr(n);
		for(size_t i =0; i<arr.size(); i++)
			arr[i] = i + 1;

		Array<int> hArr;
		hArr.construct_in_host_for_device(arr);

		hipMalloc(&dsum, sizeof(int));
		k_sum_array<<<1, 1>>>(hArr, dsum);

		hipMemcpy(&hsum, dsum, sizeof(int), hipMemcpyDeviceToHost);
		std::cout << hsum << '\n';
	return 0;
}